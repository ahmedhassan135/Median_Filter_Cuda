
#include <hip/hip_runtime.h>
#include <stdio.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Values for MxN matrix
#define M 10
#define N 10

// Kernel
__global__ void add_matrices(int *a, int *b, int *c)
{
	int column = blockDim.x * blockIdx.x + threadIdx.x;
	int row    = blockDim.y * blockIdx.y + threadIdx.y;

	int array[32];
	int count = 0;

	if (row < M && column < N)
	{
		int thread_id = row * N + column;
		//c[thread_id] = a[thread_id] + b[thread_id];
		//c[thread_id] = a[thread_id] + 1;

		if(row > 1 && column > 1 && row < M - 2 && column < N - 2)
		{
			for(int i = 0 ; i < 25; i++)
			{
				for (int j = 0 ; j < b[i]; j++)
				{
					if (i < 5)
						array[count] = a[thread_id - (2*N) - 2 + i];

					else if (i < 10)
						array[count] = a[thread_id - N - 2 + i - 5];

					else if (i < 15)
						array[count] = a[thread_id - 2 + i - 10];

					else if (i < 20)
						array[count] = a[thread_id + N - 2 + i - 15];

					else if (i < 25)
						array[count] = a[thread_id + (2*N) - 2 + i - 20];
						
					count++;
				}
			}

			
			
			


			for (int i = 0 ; i < 32; i++)
			{
				for (int j = 0 ; j < 32; j++)
				{
					if (array[j] > array [j + 1])
					{
						int temp = array[j];
						array[j] = array[j+1];
						array[j+1] = temp;
					}
				}
			}	



		
		if(row == 2 && column == 3)
		{
			for (int j = 0 ; j < 32; j++)
					{
						printf("%d ", array[j]);
							
					}


		}		




			c[thread_id] = array[31/2];

		}
		//int num = a[thread_id];
		
		

	}
}

// Main program
int main()
{
	// Number of bytes to allocate for MxN matrix
	size_t bytes = M*N*sizeof(int);

	// Allocate memory for arrays A, B, and C on host
	int A[M][N]=
    	{
		{249,255,252,255,235,0,255,255,255,243},
		{244,255,241,255,255,253,252,0,255,233},
		{255,255,242,248,249,239,248,3,237,255},
		{245,244,254,255,255,255,250,255,255,238},
		{255,241,255,242,255,236,255,0,254,251},
		{253,255,244,255,250,255,245,251,252,255},
		{233,255,248,239,255,243,255,251,4,0},
		{255,240,252,252,255,252,238,255,252,255},
		{255,248,253,247,255,252,255,247,253,255},
		{250,0,251,255,246,247,240,255,246,244}
    	};
	
		


	int C[M][N];

	

	int B[5][5] =
    	{
		{0,1,1,1,0},
		{1,2,2,2,1},
		{1,2,4,2,1},
		{1,2,2,2,1},
		{0,1,1,1,0}
    	};

	for(int i=0; i<5; i++)
	{
		for(int j=0; j<5; j++)
		{
			printf("%d", B[i][j]);
		}
		printf("\n");
	}

	

	// Allocate memory for arrays d_A, d_B, and d_C on device
	int *d_A, *d_B, *d_C;
	cudaErrorCheck( hipMalloc(&d_A, bytes) );
	cudaErrorCheck( hipMalloc(&d_B, bytes) );
	cudaErrorCheck( hipMalloc(&d_C, bytes) );

	// Initialize host arrays A and B
	for(int i=0; i<M; i++)
	{
		for(int j=0; j<N; j++)
		{
			C[i][j] = 1;
		}
	}

	

	// Copy data from host arrays A and B to device arrays d_A and d_B
	cudaErrorCheck( hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice) );
	cudaErrorCheck( hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice) );

	// Set execution configuration parameters
	// 		threads_per_block: number of CUDA threads per grid block
	//		blocks_in_grid   : number of blocks in grid
	//		(These are c structs with 3 member variables x, y, x)
	dim3 threads_per_block( 16, 16, 1 );
	dim3 blocks_in_grid( ceil( float(N) / threads_per_block.x ), ceil( float(M) / threads_per_block.y ), 1 );

	// Launch kernel
	add_matrices<<< blocks_in_grid, threads_per_block >>>(d_A, d_B, d_C);

	// Check for errors in kernel launch (e.g. invalid execution configuration paramters)
  hipError_t cuErrSync  = hipGetLastError();

	// Check for errors on the GPU after control is returned to CPU
  hipError_t cuErrAsync = hipDeviceSynchronize();

  if (cuErrSync != hipSuccess) 
	{ printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrSync)); exit(0); }

  if (cuErrAsync != hipSuccess) 
	{ printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErrAsync)); exit(0); }

	// Copy data from device array d_C to host array C
	cudaErrorCheck( hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost) );

	// Verify results
	printf("Displaying data\n\n");
	

	for(int i=0; i<M; i++)
	{
		for(int j=0; j<N; j++)
		{
			printf("%d\t", C[i][j]);
		}
		printf("\n");
	}

	// Free GPU memory
	cudaErrorCheck( hipFree(d_A) );
	cudaErrorCheck( hipFree(d_B) );
	cudaErrorCheck( hipFree(d_C) );

  printf("\n--------------------------------\n");
  printf("__SUCCESS__\n");
  printf("--------------------------------\n");
  printf("M                         = %d\n", M);
	printf("N                         = %d\n", N);
  printf("Threads Per Block (x-dim) = %d\n", threads_per_block.x);
  printf("Threads Per Block (y-dim) = %d\n", threads_per_block.y);
  printf("Blocks In Grid (x-dim)    = %d\n", blocks_in_grid.x);
	printf("Blocks In Grid (y-dim)    = %d\n", blocks_in_grid.y);
  printf("--------------------------------\n\n");

	return 0;
}
