#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include <cstdlib>
#include <cstring>
#include <stdio.h>

using namespace cv;
using namespace std;



// Values for MxN matrix Need to make it dynamic
#define M 562
#define N 800


__global__ void median_filter(int *a, int *b, int *c)
{
	int column = blockDim.x * blockIdx.x + threadIdx.x;
	int row    = blockDim.y * blockIdx.y + threadIdx.y;

	int array[32];
	int count = 0;

	if (row < M && column < N)
	{
		int thread_id = row * N + column;
		

		if(row > 1 && column > 1 && row < M - 2 && column < N - 2)
		{
			for(int i = 0 ; i < 25; i++)
			{
				for (int j = 0 ; j < b[i]; j++)
				{
					if (i < 5)
						array[count] = a[thread_id - (2*N) - 2 + i];

					else if (i < 10)
						array[count] = a[thread_id - N - 2 + i - 5];

					else if (i < 15)
						array[count] = a[thread_id - 2 + i - 10];

					else if (i < 20)
						array[count] = a[thread_id + N - 2 + i - 15];

					else if (i < 25)
						array[count] = a[thread_id + (2*N) - 2 + i - 20];
						
					count++;
				}
			}

			
			
			


			for (int i = 0 ; i < 32; i++)
			{
				for (int j = 0 ; j < 32; j++)
				{
					if (array[j] > array [j + 1])
					{
						int temp = array[j];
						array[j] = array[j+1];
						array[j+1] = temp;
					}
				}
			}	



			
			c[thread_id] = array[31/2];

		}
		//int num = a[thread_id];

	}
}


int main( int argc, char** argv )
{
	
	size_t bytes = M*N*sizeof(int);

	int A[M][N];		//Image array

	for (int i=0; i < M;i++)
	{
		for (int j = 0; j < N; j++)
		{
			A[i][j] = -1;
		}
	}


	int C[M][N];		//Result array

	

	int B[5][5] =					//5x5 Image Filter kernel
    	{
		{0,1,1,1,0},
		{1,2,2,2,1},
		{1,2,4,2,1},
		{1,2,2,2,1},
		{0,1,1,1,0}
    	};

	//cout<<"printing data\n";
    if( argc != 2)
    {
     cout <<" Usage: display_image ImageToLoadAndDisplay" << endl;
     return -1;
    }

    Mat image;
    image = imread(argv[1]);   // Read the image

    if(! image.data )                              // Check for invalid input
    {
        cout <<  "Could not open or find the image" << std::endl ;
        return -1;
    }

	
	
	for(int j=0;j<M;j++) 
	{
	  for (int i=0;i<N;i++)
	  {
		  /*the opencv mat image stores the image in this format (10,10,10) (0,0,0) (5,5,5),
		   same value across RGB except this is grayscale so 3 same values per pixel which makes
		   the image columns 3 times larger since they store 3 same values to retrieve unique
		   values 'i' index here is multiplied by 3 each time*/
	       A[j][i] = (int)image.at<uchar>(j,i*3);
		//count++;
	  }
		//cout<<"\n";
	}

	



	//CUDA function call here
	

	int *d_A, *d_B, *d_C;
	hipMalloc(&d_A, bytes);
	hipMalloc(&d_B, bytes);
	hipMalloc(&d_C, bytes);

	
	for(int i=0; i<M; i++)
	{
		for(int j=0; j<N; j++)
		{
			C[i][j] = 1;
		}
	}



	hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

	
	dim3 threads_per_block( 16, 32, 1 );
	dim3 blocks_in_grid( ceil( (float(N) / threads_per_block.x) ), ceil( float(M) / threads_per_block.y ), 1 );

	// Launch kernel
	median_filter<<< blocks_in_grid, threads_per_block >>>(d_A, d_B, d_C);

	

 
	
	hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

	
	//printf("Displaying data\n\n");
	
	int count = 0;


	for(int j=0;j<M;j++) 
	{		
	  for (int i=0;i<N;i++)
	  {
		  /*Since cv image is stored redundantly in grayscale, this is done to store same value 3
		  times*/
		   image.at<uchar>(j,count) = C[j][i];
		   image.at<uchar>(j,count + 1) = C[j][i];
		   image.at<uchar>(j,count + 2) = C[j][i];
		   count = count + 3;
		//count++;
	  }
	  count = 0;
		//cout<<"\n";
	}

	

	// Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);


	
	

    namedWindow( "Display window", WINDOW_AUTOSIZE );
    imshow( "Display window", image ); 
  //imwrite("Denoised_image", image);                 
   
    waitKey(0);                                       


	
	

    return 0;
}
